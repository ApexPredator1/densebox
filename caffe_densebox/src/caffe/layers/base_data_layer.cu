#include <vector>

#include "caffe/layers/base_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(batch->label_);
    // Copy the labels.
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);







template <typename Dtype>
void OLD_BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // First, join the thread
  JoinPrefetchThread();
  // Reshape to loaded data.
  top[0]->Reshape(this->prefetch_data_.num(), this->prefetch_data_.channels(),
      this->prefetch_data_.height(), this->prefetch_data_.width());
  // Copy the data
  caffe_copy(prefetch_data_.count(), prefetch_data_.cpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    caffe_copy(prefetch_label_.count(), prefetch_label_.cpu_data(),
        top[1]->mutable_gpu_data());
  }
  // Start a new prefetch thread
  CreatePrefetchThread();
}

INSTANTIATE_LAYER_GPU_FORWARD(OLD_BasePrefetchingDataLayer);


}  // namespace caffe
